#include "hip/hip_runtime.h"
// mepidemic.cu: CUDA kernels for determining the number of transmission events that should occur
// to the mosquitoes of each subpopulation and infecting mosquitoess.

#include "hiprand/hiprand_kernel.h"              // Required for random number generation in CUDA.
#include "cudaidentities.h"             // Streaming multiprocessor ID, warp ID, and lane ID.
#include ""   // Thread ID and block ID.
#include "censustypedef.h"              // Type definitions for census data.
#include "Parameter.h"                  // Parameter space definition.
#include "constant.h"                   // Constants for simulation.

// Device function which determines if a float should be rounded up or down, dependent
// on uniform random number generation.
__device__ __forceinline__ uint32_t decimalResolve(hiprandState_t* state, float floating)
{
    float frac = floating - floorf(floating);
    if (hiprand_uniform(state) <= frac)
    {
        return static_cast<uint32_t>(ceilf(floating));
    }
    else
    {
        return static_cast<uint32_t>(floorf(floating));
    }
}

// CUDA kernel which determines the number of transmission events that should occur to
// mosquitoes in every subpopulation in a system where the human to mosquito population ratio is 1 to 1.
__global__ void mVisitingInfected(hiprandState_t* d_randStates,
                                  uint32_t* d_mTransmission,
                                  const uint32_t* d_infectedCount,
                                  const float* d_sparseNetWeight,
                                  const uint32_t* d_sparseNetTo,
                                  const uint32_t* d_sparseNetLoc,
                                  const uint32_t* d_subPopTotal,
                                  const Parameter h_parameter)
{
    // Initialize the strain, subpopulation combination of the thread.
    uint32_t strainSubPop = blockIdx.x*blockDim.x + threadIdx.x;

    // Load in the total number of subpopulation from global memory to L1 cache.
    uint32_t subPopTotal = __ldg(&d_subPopTotal[0]);

    // Check if the strain, subpopulation combination is within limits.
    if (strainSubPop < subPopTotal*C_STRAINS)
    {
        // Load the number of infected mosquitoes in that subpopulation, strain combination.
        uint32_t infectedCount = __ldg(&d_infectedCount[strainSubPop]);

        // If there are infected individuals, then compute the number of transmission events
        // that they would cause.
        if (infectedCount > 0)
        {
            // Determine the source subpopulation of the thread.
            uint32_t fromSubPop = strainSubPop % subPopTotal;

            // Initialize the active thread index used for random number generation.
            uint32_t activeThreadId = smId()*h_parameter.arch.warpsPerSM*h_parameter.arch.threadsPerWarp
                + warpId()*h_parameter.arch.threadsPerWarp + laneId();

            // Load in the random number generator into local memory.
            hiprandState_t local_state = d_randStates[activeThreadId];

            // Determine the total number of transmission events that the infected humans would cause 
            // on a mosquito population of the same size as the human population.
            uint32_t transmissions = decimalResolve(&local_state,
                static_cast<float>(h_parameter.simulation.bitingRate*h_parameter.simulation.mnBitingSuccess*infectedCount));

            // Delcare a variable for how many transmission events are not dispersed to a different community.
            uint32_t noDistanceTransmissions;

            // Initialize the location in global memory where transmission numbers to individuals
            // of every subpopulation begins for each strain.
            uint32_t strainStartIndex = subPopTotal*(strainSubPop / subPopTotal);

            // Get the location in the network weight array where the source subpopulations
            // non-zero weights begin. 
            uint32_t sparseNetLoc = d_sparseNetLoc[fromSubPop];

            // Get the number of connections that the source subpopulation has.
            uint32_t connections = d_sparseNetLoc[fromSubPop + 1] - sparseNetLoc;

            // Only send transmission events to other communities if the source community is not isolated.
            if (connections > 0)
            {
                // Compute the number of transmission events that remain in the source community.
                noDistanceTransmissions = decimalResolve(&local_state, transmissions*h_parameter.simulation.noDistance);
                transmissions -= noDistanceTransmissions;

                // Calculate the total number of long-distance transmissions to random communities.
                uint32_t forcedLongDistance = decimalResolve(&local_state, transmissions*h_parameter.simulation.longDistance);
                transmissions -= forcedLongDistance;

                // Declare the subpopulation that each transmission event is sent to.
                uint32_t toSubPop;

                // Provided there are still local transmission numbers to be made.
                while (transmissions > 0)
                {
                    // Generate a random number between [0 and connections - 1].
                    toSubPop = static_cast<uint32_t>(hiprand(&local_state) % connections);

                    // In the current revision of the code, we know the weight is 1, so
                    // just get the destination subpopulation, or column index, in the sparse matrix.
                    toSubPop = d_sparseNetTo[sparseNetLoc + toSubPop];

                    // Increment the number of transmissions that should be placed upon individuals
                    // of the destination subpopulation.
                    atomicAdd(&d_mTransmission[strainStartIndex + toSubPop], 1);

                    // Decrease the number of local transmission events to disperse.
                    --transmissions;
                }

                // Provided there are long distance transmission events to make, randomly choose
                // a subpopulation within the lattice, and disperse the transmission event to the
                // individuals of that subpopulation.
                while (forcedLongDistance > 0)
                {
                    toSubPop = static_cast<uint32_t>(hiprand(&local_state) % subPopTotal);
                    atomicAdd(&d_mTransmission[strainStartIndex + toSubPop], 1);
                    --forcedLongDistance;
                }
            }          
            else
            {
                // If the source community has no neighbours, all transmission events remain within that
                // community.
                noDistanceTransmissions = transmissions;
            }

            // Add on the number of non-dispersed transmission events to the source community.
            atomicAdd(&d_mTransmission[strainStartIndex + fromSubPop], noDistanceTransmissions);

            // Store the random number generator state back to global memory.
            d_randStates[activeThreadId] = local_state;
        }
    }
}

// CUDA kernel which infects the mosquitoes of every subpopulation given the number
// of transmission events that should occur in a subpopulation with a one to one human
// to mosquito population ratio.
__global__ void nmTransmission(age* d_mAge,
                               dead* d_mDead,
                               exposed* d_mExposed,
                               infectStatus* d_mInfectStatus,
                               strain* d_mStrain,
                               hiprandState_t* d_randStates,
                               const uint32_t* d_subPopIndex,
                               const uint32_t* d_subPopLoc,
                               const uint32_t* d_subPopSize,
                               const uint32_t* d_nmTransmission,
                               const uint32_t* d_mSubPopCount,
                               const uint32_t* d_size,
                               const uint32_t* d_subPopTotal,
                               const uint32_t t,
                               const Parameter h_parameter)
{
    // Initialize the subpopulation of the thread.
    uint32_t subPop = blockIdx.x*blockDim.x + threadIdx.x;

    // Load in the total number of subpopulation from global memory to L1 cache.
    uint32_t subPopTotal = __ldg(&d_subPopTotal[0]);

    // Initialize the active thread index used for random number generation.
    uint32_t activeThreadId = smId()*h_parameter.arch.warpsPerSM*h_parameter.arch.threadsPerWarp
        + warpId()*h_parameter.arch.threadsPerWarp + laneId();

    // Check if the subpopulation of the thread is within limits.
    if (subPop < subPopTotal)
    {
        // Load in the maximum number of individuals in the subpopulation.
        uint32_t subPopSize = __ldg(&d_subPopSize[subPop]);

        // Load in the total number of alive individuals in the subpopulation.
        uint32_t subPopCount = __ldg(&d_mSubPopCount[subPop]);

        // Check that individuals can exist in the subpopulation
        if (subPopCount > 0)
        {
            // Read in the random number generator state into global memory.
            hiprandState_t local_state = d_randStates[activeThreadId];

            // Load in the location where to begin in the sub-population ordered census
            // indices for the human population for the subpopulation of the thread.
            uint32_t subPopLoc = __ldg(&d_subPopLoc[subPop]);

            // Load in the total number of humans alive in the entire population.
            uint32_t local_size = __ldg(&d_size[0]);

            // Randomly choose a strain of the virus to begin infecting individuals of the 
            // subpopulation with.
            uint32_t local_strain = hiprand(&local_state) % C_STRAINS;

            // Cycle through all the strains, infecting humans give the total number of transmission events that
            // occur to individuals of the subpopulation per strain.
            for (uint32_t strainCount = 0; strainCount < C_STRAINS; ++strainCount)
            {
                // Load in the number of transmission events of a one to one mosquito to human population ratio,
                // and multiply by the actual mosquito to human ratio to get the total number of transmission events
                // that act on mosquitoes in the subpopulation.
                uint32_t transmissionCount = decimalResolve(&local_state, 
                    static_cast<float>(d_nmTransmission[local_strain*subPopTotal + subPop]));

                // Provided there are transmission events:
                while (transmissionCount > 0)
                {
                    // Choose a random individual in the sub-population.
                    uint32_t censusIndex = hiprand(&local_state) % subPopSize;

                    // Find their index within the census data by using the sub-population ordered
                    // census indices.
                    censusIndex = d_subPopIndex[subPopLoc + censusIndex];

                    // If their census index is valid (which it should be by construction anyway), 
                    // and the mosquito is actually alive, continue with transmission.
                    if ((censusIndex < local_size) && (d_mDead[censusIndex] == 0))
                    {
                        // The virus is transmitted to the individual, so decrease the number of 
                        // remaining transmission events to make.
                        --transmissionCount;

                        // The individual will be infected if it is not already infected.
                        if (d_mInfectStatus[censusIndex] == 0)
                        {
                            // Infected, not infectious.
                            d_mInfectStatus[censusIndex] = 1;
                            d_mStrain[censusIndex] = local_strain;

                            // Age at which individual becomes infectious.
                            d_mExposed[censusIndex] = static_cast<exposed>(decimalResolve(&local_state,
                                static_cast<float>(d_mAge[censusIndex] + h_parameter.simulation.mExposed)));
                        }
                    }
                }        

                // Move onto the transmission events of the next strain.
                local_strain = (local_strain + 1) % C_STRAINS;
            }

            // Store the random number generator back to global memory.
            d_randStates[activeThreadId] = local_state;

        }
    }
}